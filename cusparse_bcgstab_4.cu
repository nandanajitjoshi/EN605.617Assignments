#include "hip/hip_runtime.h"
/*Matrix A - define in CSR, convert to BSR*/
/*Vector b, vector r, Vector r-o*/
/*rho, alphs, pmega, itr - Scalars*/
//rho, i-1 and rho_i, omega_i-1 and omega_i
//Steps 
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include "helper_functions.h"  // helper for shared functions common to CUDA Samples
#include "hip/hip_runtime_api.h"       // helper function CUDA error checking and initialization
/*Initialize CSR*/

void genTridiag(int *I, int *J, double*val, int M, int N)
{

    I[0] = 1;
    int start = 0; 

    for (int i = 1; i <= M*N; i++)
    {
        start = (i-1)*5;
        
        if ( i <= N){
            J[start + 4] = i+N*(M-1);
            J[start] = i-1;
            J[start+1] = i;
            J[start+2] = i+1;
            J[start + 3] = i+N;

            val[start + 4] = 0;
            val[start] = 0;
            val[start+1] = 1;
            val[start+2] = 0;
            val[start + 3] = 0;


            if (i%N ==1){
            J[start + 4] = i+N*(M-1);
            J[start] = i;
            J[start+1] = i+1;
            J[start+2] = i+N-1;
            J[start + 3] = i+N;

            val[start + 4] = 0;
            val[start] = 1;
            val[start+1] = 0;
            val[start+2] = 0;
            val[start + 3] = 0;
            
            }else if ((i%N==0)) {
                J[start + 4] = i+N*(M-1);
                J[start] = i-N+1;
                J[start+1] = i-1;
                J[start+2] = i;
                J[start + 3] = i+N;

                val[start + 3] = 0;
                val[start+4] = 0;
                val[start] = 0;
                val[start+1] = 0;
                val[start + 2] = 1;
            
            }
            
        } else if (i > (M-1)*N){

            J[start] = i-(M-1)*N;
            J[start+1] = i-N;
            J[start+2] = i-1;
            J[start+3] = i;
            J[start + 4] = i+1;

            val[start + 4] = 0;
            val[start] = 0;
            val[start+1] = 0;
            val[start+3] = 1;
            val[start + 2] = 0;

            if (i%N ==1 ){
                J[start] = i-(M-1)*N;
                J[start+1] = i-N;
                J[start+2] =i;
                J[start+3] = i+1;
                J[start + 4] = i+N-1;

                val[start + 4] = 0;
                val[start] = 0;
                val[start+1] = 0;
                val[start+3] = 0;
                val[start + 2] = 1;
            } else if (i%N==0){
                J[start] = i-(M-1)*N;
                J[start+1] = i-N;
                J[start+2] =i-N+1;
                J[start+3] = i-1;
                J[start + 4] = i;

                val[start + 4] = 1;
                val[start] = 0;
                val[start+1] = 0;
                val[start+3] = 0;
                val[start + 2] = 0;
            
            }
            


        } else if ( i%N == 1 ){
                J[start + 3] = i+N-1;
                J[start+4] = i+N;
                J[start] = i-N;
                J[start+1] = i;
                J[start + 2] = i+1;

                val[start + 3] = 0;
                val[start+4] = 0;
                val[start] = 0;
                val[start+1] = 1;
                val[start + 2] = 0;

        } else if ((i%N==0)){
                J[start + 3] = i;
                J[start+4] = i+N;
                J[start] = i-N;
                J[start+1] = i-N+1;
                J[start + 2] = i-1;

                val[start + 3] = 1;
                val[start+4] = 0;
                val[start] = 0;
                val[start+1] = 0;
                val[start + 2] = 0;
        } else {
            J[start + 4] = i+N;
            J[start] = i-N;
            J[start+1] = i-1;
            J[start+2] = i;
            J[start + 3] = i+1;


            val[start + 4] = 1;
            val[start] = 1;
            val[start+1] = 1;
            val[start+2] = -4;
            val[start + 3] = 1;
            
        }

          I[i] = 5 + I[i-1]; 
    }
}

void* getBSRDims(hipsparseHandle_t handle, int* rowPtr, int* colPtr, double* val, 
        int* rowBSR, int*nnzb, 
        int rows,  int dimBlock
        ){


    hipsparseMatDescr_t descr_coeff;
    hipsparseMatDescr_t descr_coeff_2;
    
    int bufferSize = 0; 
    static void *pBuffer; 

    const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff_2));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff_2, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff_2, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseDcsr2gebsr_bufferSize(handle, dir_coeff, rows, rows,  descr_coeff, 
    val, rowPtr, colPtr, dimBlock, dimBlock, &bufferSize));

    checkCudaErrors(hipMalloc((void**)&pBuffer, bufferSize));

    checkCudaErrors(hipsparseXcsr2gebsrNnz(handle, dir_coeff, rows, rows, descr_coeff, rowPtr, colPtr,
    descr_coeff_2, rowBSR, dimBlock, dimBlock, nnzb, pBuffer));

    return pBuffer; 

}

void BCGSolve(double* X, double* RHS, int* rowBSR, int* colBSR, double*valBSR, 
        hipsparseHandle_t handle,  hipsparseMatDescr_t descr_coeff, 
        hipblasHandle_t handleBlas, 
        int mb, int nb, int nnzb, int vecSize, int maxit  ){

        double *R, *V, *T, *rw, *p;
        int dimBlock = 2; 

        double alpha = 1 ;
        double beta ;
        double alph;
        double bet;
        double omega = 1;
        double residual1 = 0;
        double residual2 = 0;
        double temp = 1;
        double rhop = 1; 
        double rho = 1;

        checkCudaErrors(hipMalloc((double **)&R, (vecSize)*sizeof(double)));
        checkCudaErrors(hipMalloc((double **)&p, (vecSize)*sizeof(double)));
        checkCudaErrors(hipMalloc((double **)&rw, (vecSize)*sizeof(double)));
        checkCudaErrors(hipMalloc((double **)&V, (vecSize)*sizeof(double)));
        checkCudaErrors(hipMalloc((double **)&T, (vecSize)*sizeof(double)));

        checkCudaErrors(hipMemset(V, 0, vecSize)); 


        const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

        double* Y = (double*)malloc(vecSize*sizeof(double)); 


        /*r = b - r [Ax]*/ 
        alph = -1;
        bet = 0;
   
        // printf ("Entered here");

        checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
                                      descr_coeff,valBSR, rowBSR, colBSR, dimBlock,X, 
                                      &bet, R));


        alph = 1;  
        checkCudaErrors( hipblasDaxpy( handleBlas, vecSize, 
                                     &alph,RHS, 1,R, 1));

        checkCudaErrors (hipMemcpy(Y, R,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

        // printf ("\n R matrix:\n");
        // for (int i = 0; i < vecSize; i++){
            
        //     printf ("%f\n", Y[i]);       

        // }

    




    //2: Set p=r and \tilde{r}=r
    checkCudaErrors(hipblasDcopy(handleBlas,(vecSize), R, 1, p, 1));
    checkCudaErrors(hipblasDcopy(handleBlas, (vecSize), R, 1, rw,1));
    checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual1));  //Changed from row*cols to vecSize
    printf (" Residual %f \n", residual1);


    //3: repeat until convergence (based on max. it. and relative residual)
    for (int i=0; i<maxit; i++){
        rhop = rho; 
        //Step 5.1 : Dot product (rw,r)
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, rw, 1, R, 1, &rho));
        // printf ("\n Dot Product %f \n", rho);
        printf("Iteration %d \n",i);

        if (i > 0){
            //5.2: \beta = (\rho_{i} / \rho_{i-1}) ( \alpha / \omega )
            beta= (rho/rhop)*(alpha/omega);
            printf ("\n Beta %f \n", beta);

            //5.3: p = r + \beta (p - \omega v)

            //-omega*v
            omega = -omega; 
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                            &omega,V, 1,p, 1));

            //Reset omega
            omega = -omega;


            //beta * (p - omega*v)
            checkCudaErrors(hipblasDscal(handleBlas, vecSize, 
                                &beta,p, 1)); 

            // r + beta*(p-omega*v)
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                        &alph,R, 1,p, 1)); 


            checkCudaErrors (hipMemcpy(Y, p,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

            // printf ("\n Updated P  matrix:\n");
            // for (int i = 0; i < vecSize; i++){
            
            //     printf ("%f\n", Y[i]);       

            // }
        }
            
            //Step 5.4 : v = A*p
            checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
                descr_coeff,valBSR, rowBSR, colBSR, dimBlock,p, 
                &bet, V));


            checkCudaErrors (hipMemcpy(Y, V,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

            // printf ("\n V matrix:\n");
            // for (int i = 0; i < vecSize; i++){
                
            //     printf ("%f\n", Y[i]);       

            // }

            //Step 5.5 : alpha = rho_i/(r_tilde * v_i)

            // alpha = (r_tilde * v_i)
            checkCudaErrors(hipblasDdot ( handleBlas, vecSize, rw, 1, V, 1, &alpha));

            // printf ("\n Alpha %f \n", alpha);

            //alpha = rho/alpha
            alpha = rho/alpha; 

            // printf ("\n Alpha 2 %f \n", alpha);

            /*Step 5.6/ s = r - \alpha * v */
            alpha = -alpha; 
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                        &alpha,V, 1,R, 1));

            checkCudaErrors (hipMemcpy(Y, R,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

            // printf ("\n Updated R matrix:\n");
            // for (int i = 0; i < vecSize; i++){
                
            //     printf ("%f\n", Y[i]);       

            // }

            //Reset alpha
            alpha = -alpha; 

            /*Step 5.5 Y = Y + p*alpha*/
            checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                &alpha,p, 1,X, 1));

            /*Step 5.7 : Check the residual of s (r)*/
            checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual2));

            if (residual2/residual1 < 1E-3){
                break;
            }

        
        /*Step 5.9 T = A*s(r)*/
        checkCudaErrors(hipsparseDbsrmv(handle, dir_coeff, trans_coeff, mb, nb, nnzb, &alph,
            descr_coeff,valBSR, rowBSR, colBSR, dimBlock,R, 
            &bet, T));


        checkCudaErrors (hipMemcpy(Y, T,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

        // printf ("\n Updated T matrix:\n");
        // for (int i = 0; i < vecSize; i++){
            
        //     printf ("%f\n", Y[i]);       

        // }

        /*Step 5.10 omega = (T.T)/(T.R)*/
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, T, 1, T, 1, &temp));  //Changed from row*cols to vecSize
        checkCudaErrors(hipblasDdot ( handleBlas, vecSize, R, 1, T, 1, &omega));  //Changed from row*cols to vecSize

        // printf ("\n Omega %f \n", omega);
        omega = omega/temp; 
        // printf ("\n Omega 2%f \n", omega);

        //Step 5.11 *x = h + omega *s*/  

        checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                    &omega,R, 1,X, 1));


        /*Step 5.13 r = s - omega * t*/

        omega = -omega; 
        checkCudaErrors(hipblasDaxpy(handleBlas, vecSize, 
                    &omega,T, 1,R, 1));
        //Reset omega
        omega = -omega; 

        checkCudaErrors (hipMemcpy(Y, R,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

        // printf ("\n Updated Final Residual matrix:\n");
        // for (int i = 0; i < vecSize; i++){
            
        //     printf ("%f\n", Y[i]);       

        // }

        /*Step 5.12 Check residual of R*/
        checkCudaErrors(hipblasDnrm2(handleBlas,(vecSize), R, 1, &residual2));

         printf ("\n Residual %f \n", residual2/residual1);

        if (residual2/residual1 < 1E-3){
            break;
        }
    }


}


void LinearSolve( int* rowPtr, int* colPtr, double* val, 
                  double* Soln, double* RHS,                   
                  int rows, int nz, int maxit ){
    //double*val_host = NULL;
    //double*val = NULL;
    double*val_BSR = NULL;
    //int* row_host, *row, *row_BSR, *col_host, *col, *col_BSR; 
    double* d_Y, *d_X; 
    int* row_BSR, *col_BSR; 
    int dimBlock = 2;


   

    int nRows = rows+1;
    int mb = (rows + dimBlock-1)/dimBlock;
    int nb = (rows + dimBlock-1)/dimBlock; 
    int base; 
    int vecSize = rows + dimBlock-1;

    checkCudaErrors(hipMalloc((double **)&d_X, (vecSize)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&d_Y, (vecSize)*sizeof(double)));
    double* host = (double*)malloc(vecSize*sizeof(double)); 



    int bufferSize;
    void *pBuffer;
    int nnzb = 0; 
  

    hipsparseHandle_t handle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&handle);
    
    hipblasHandle_t handleBlas = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&handleBlas);

    const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW;

    hipsparseMatDescr_t descr_coeff;
    hipsparseMatDescr_t descr_coeff_2;
    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));

    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff_2));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff_2, HIPSPARSE_INDEX_BASE_ZERO));
    checkCudaErrors(hipsparseSetMatType(descr_coeff_2, HIPSPARSE_MATRIX_TYPE_GENERAL));


    checkCudaErrors(hipMalloc((void**)&row_BSR, sizeof(int) *(mb+1)));



    pBuffer =  getBSRDims(handle, rowPtr, colPtr, val, row_BSR, &nnzb,
                    rows, dimBlock); 

    // printf("%d\n",nnzb);
    // printf("%d\n", nz);


    checkCudaErrors(hipMalloc((void**)&col_BSR, sizeof(int)*(nnzb)));
    checkCudaErrors(hipMalloc((void**)&val_BSR, sizeof(double)*(dimBlock*dimBlock)*(nnzb)));

    checkCudaErrors(hipsparseDcsr2gebsr(handle, dir_coeff,rows, rows, descr_coeff, 
       val, rowPtr, colPtr, descr_coeff_2, val_BSR, row_BSR, col_BSR,        
        dimBlock, dimBlock, pBuffer));

    checkCudaErrors(hipblasDcopy(handleBlas,rows, RHS, 1, d_X, 1));
    checkCudaErrors(hipblasDcopy(handleBlas, rows, Soln, 1, d_Y,1));

    checkCudaErrors (hipMemcpy(host, d_X,  (vecSize)*sizeof(double), hipMemcpyDeviceToHost ));

    // printf ("\n R matrix:\n");
    // for (int i = 0; i < vecSize; i++){
            
    //     printf ("%f\n", host[i]);       

    // }



    BCGSolve(d_Y,  d_X,  row_BSR,  col_BSR, val_BSR, 
              handle, descr_coeff_2, handleBlas, 
               mb, nb, nnzb, vecSize, maxit); 

    checkCudaErrors(hipblasDcopy(handleBlas, rows, d_Y, 1, Soln,1));

}
/*Convert CSR to BSR*/
/*Initialize x, b, ro and r_i*/
/*rho_i = dotproduct (r0',ri-1)*/
/*Beta = rho_i/rho_iminusone * alphs/omegaiminusone*/
/*rho_i = r_i+beta*(p_i-1 - omega_i-1*v_iminusone)*/
/*Matrix vector multiply - A and rho_i*/
/*alpha = rho_i/(r_o',vi) - Scalar dot prodyct*/
/*h = x + alpha * p_i*/
/*check residual b/w h and x*/
/*s = r_iinusone - alpha*v_i*/
/*t = A*s - Matrix vector multiply*/
/*omega_i = dot(t,s)/dot(t,t)*/
/*x_i = h + omega_i*s*/
/*r = s-omega*t*/