#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. AN use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/*
 * This sample implements a conjugate gradient solver on GPU
 * using CUBLAS and CUSPARSE
 *
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include "helper_functions.h"  // helper for shared functions common to CUDA Samples
#include "hip/hip_runtime_api.h"       // helper function CUDA error checking and initialization

const char *sSDKname     = "conjugateGradient";

/* genTridiag: generate a random tridiagonal symmetric matrix */
void genTridiag(int *I, int *J, double*val, int M, int N)
{

    I[0] = 1;
    int start = 0; 

    for (int i = 1; i <= M*N; i++)
    {


        start = (i-1)*5;
        J[start] = i - N;
        J[start+1] = i-1;
        J[start+2] = i;
        J[start+3] = i+1;
        J[start + 4] = i+N; 

        val[start] = -1;
        val[start+1] = -1;
        val[start+2] = 4;
        val[start+3] = -1;
        val[start+4] = -1;


        if ( i < N){
            J[start + 4] = M*N-N+i;
            J[start] = i-1;
            J[start+1] = i;
            J[start+2] = i+1;
            J[start + 3] = i+N;

            val[start+1] = 4;
            val[start+2] = -1;


            if (i==1){
                J[start] = i;
                J[start+1] = i+1;
                J[start + 2] = i+N-1;

                val[start] = 4;
                val[start+1] = -1;
            }
        }

        if ( i > (M-1)*N){
            J[start + 1] = i-N;
            J[start+2] = i-1;
            J[start+3] = i;
            J[start+4] = i+1;
            J[start] = N - (N*M-i);

            val[start+3] = 4;
            val[start+2] = -1;

            if (i==M*N){
                J[start + 2] = M*N-N+1;
                J[start+3] = i-1;
                J[start+4] = i;

                val[start+4] = 4;
                val[start+3] = -1;
            }
        }

        if ((i%N==1)&& (i!=1)){
            J[start] = i - N;
            J[start+1] = i;
            J[start+2] = i+1;
            J[start+3] = i+N-1;
            J[start + 4] = i+N;

            val[start+1] = 4;
            val[start+2] = -1;

            if (i==((M*N)-N+1)){

                J[start] = 1;
                J[start+1] = i-N;
                J[start+2] = i;
                J[start+3] = i+1;
                J[start + 4] = N*M;
                
            }    


            
            
        }

        if ((i%(N)==0)&& (i!=M*N)){
            J[start] = i - N;
            J[start+1] = i-N+1;
            J[start+2] = i-1;
            J[start+3] = i;
            J[start + 4] = i+N; 

            val[start+3] = 4;
            val[start+2] = -1;


            if (i==N){
                J[start] = 1;
                J[start+1] = i-1;
                J[start+2] = i;
                J[start+3] = i+N;
                J[start + 4] = N*M;
            }
        }

        I[i] = 5 + I[i-1]; 

      
    }


}

int main(int argc, char **argv)
{
    int M = 0, N = 0, nz = 0, *I = NULL, *J = NULL;
    double*val = NULL;
    int *d_col, *d_row;
    double*d_val, *d_x, dot;
    double*d_r, *d_p, *d_Ax;
    int k;
    double alpha, beta, alpham1;

    // This will pick the best possible CUDA capable device
    hipDeviceProp_t deviceProp;
    int devID = findCudaDevice(argc, (const char **)argv);

    if (devID < 0)
    {
        printf("exiting...\n");
        exit(EXIT_SUCCESS);
    }

    checkCudaErrors(hipGetDeviceProperties(&deviceProp, devID));

    // Statistics about the GPU device
    printf("> GPU device has %d Multi-Processors, SM %d.%d compute capabilities\n\n",
           deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);

    int version = (deviceProp.major * 0x10 + deviceProp.minor);

    if (version < 0x11)
    {
        printf("%s: requires a minimum CUDA compute 1.1 capability\n", sSDKname);

        // hipDeviceReset causes the driver to clean up all state. While
        // not mandatory in normal operation, it is good practice.  It is also
        // needed to ensure correct operation when the application is being
        // profiled. Calling hipDeviceReset causes all profile data to be
        // flushed before the application exits
        hipDeviceReset();
        exit(EXIT_SUCCESS);
    }

    /* Generate a random tridiagonal symmetric matrix in CSR format */
    M = N = 4;
    
    I = (int *)malloc(sizeof(int)*(M+1));
    J = (int *)malloc(sizeof(int)*6);
    val = (double*)malloc(sizeof(double)*6);
    //genTridiag(I, J, val, N,M);

    // printf ("\n Matrix:\n");

    // for (int i = 0; i < N*M; i++){
    //     for (int j = 0; j< 5; j++){
    //         printf ("%f\t", val[i*5+j]);
    //     }

    //     printf("\n");
    // }

    
    // printf ("\n Columns:\n");

    // for (int i = 0; i < N*M; i++){
    //     for (int j = 0; j< 5; j++){
    //         printf ("%d\t", J[i*5+j]);
    //     }

    //     printf("\n");
    // }

    // printf ("\n Rows:\n");

    // for (int i = 0; i < N*M; i++){
        
    //     printf ("%d\n", I[i]);       

    // }


    val[0] = 5;
    val[1] = 8;
    val[2] = 3;
    val[3] = 1;
    val[4] = 3;
    val[5] = 6;

    J[0] = 1;
    J[1] = 2;
    J[2] = 4;
    J[3] = 1; 
    J[4] = 4;
    J[5] = 3; 

    I[0] = 1;
    I[1] = 2;
    I[2] = 4;
    I[3] = 6;
    I[4] = 7;
 


    // x = (double*)malloc(sizeof(float)*N);
    // rhs = (double*)malloc(sizeof(float)*N);

    // for (int i = 0; i < N; i++)
    // {
    //     rhs[i] = 1.0;
    //     x[i] = 0.0;
    // }

    /* Get handle to the CUBLAS context */
    hipblasHandle_t cublasHandle = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&cublasHandle);

    checkCudaErrors(hipblasStatus_t);

    /* Get handle to the CUSPARSE context */
    hipsparseHandle_t handle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&handle);


    //define matrix variables
    hipsparseMatDescr_t descr_coeff = 0;
    hipsparseMatDescr_t descr_Lc = 0;
    hipsparseMatDescr_t descr_Uc = 0;

    csrilu02Info_t info_coeff = 0;
    csric02Info_t info_coeff_2 = 0;
    // csrsv2Info_t   info_Lc = 0;
    // csrsv2Info_t   info_Uc = 0;
    int pBufferSize_coeff;
    int pBufferSize_Lc;
    int pBufferSize_Uc;
    int pBufferSize;
    void *pBuffer = 0;
    int structural_zero;
    int numerical_zero;

    const hipsparseSolvePolicy_t policy_coeff = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_Lc = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    const hipsparseSolvePolicy_t policy_Uc = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    const hipsparseOperation_t trans_Lc  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseOperation_t trans_Uc  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    const hipsparseDirection_t dir = HIPSPARSE_DIRECTION_COLUMN;


    checkCudaErrors(cusparseStatus);

    checkCudaErrors(hipMalloc((int **)&d_col, 6*sizeof(int)));
    checkCudaErrors(hipMalloc((int **)&d_row, 5*sizeof(int)));
    checkCudaErrors(hipMalloc((double **)&d_val, 6*sizeof(double)));
    // checkCudaErrors(hipMalloc((void **)&d_x, N*sizeof(float)));
    // checkCudaErrors(hipMalloc((void **)&d_r, N*sizeof(float)));
    // checkCudaErrors(hipMalloc((void **)&d_p, N*sizeof(float)));
    // checkCudaErrors(hipMalloc((void **)&d_Ax, N*sizeof(float)));

    checkCudaErrors(hipMemcpy(d_col, J, 6*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_row, I, 5*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_val, val,6*sizeof(double), hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice));
    // checkCudaErrors(hipMemcpy(d_r, rhs, N*sizeof(float), hipMemcpyHostToDevice));
    

    printf ("\n Matrix:\n");

    for (int i = 0; i< 6; i++){
        printf ("%f\t", val[i]);
    }



    
    printf ("\n Columns:\n");

    for (int i = 0; i < 6; i++){
        printf ("%d\t", J[i]);
    }

 
     printf ("\n Rows:\n");

    for (int i = 0; i < 5; i++){
        printf ("%d\t", I[i]);
    }


    // step 1: create a descriptor which contains
    // - matrix M is base-1
    // - matrix L is base-1
    // - matrix L is lower triangular
    // - matrix L has non-unit diagonal
    checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ONE));
    checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));

    //checkCudaErrors(cusparseStatus);


    // hipsparseCreateMatDescr(&descr_Lc);
    // hipsparseSetMatIndexBase(descr_Lc, HIPSPARSE_INDEX_BASE_ONE);
    // hipsparseSetMatType(descr_Lc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    // hipsparseSetMatFillMode(descr_Lc, HIPSPARSE_FILL_MODE_LOWER);
    // hipsparseSetMatDiagType(descr_Lc, HIPSPARSE_DIAG_TYPE_NON_UNIT);


    // hipsparseCreateMatDescr(&descr_Uc);
    // hipsparseSetMatIndexBase(descr_Uc, HIPSPARSE_INDEX_BASE_ONE);
    // hipsparseSetMatType(descr_Uc, HIPSPARSE_MATRIX_TYPE_GENERAL);
    // hipsparseSetMatFillMode(descr_Uc, HIPSPARSE_FILL_MODE_UPPER);
    // hipsparseSetMatDiagType(descr_Uc, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // step 2: create a empty info structure
    // we need one info for csric02 and two info's for csrsv2
    checkCudaErrors(hipsparseCreateCsric02Info(&info_coeff_2));
    // hipsparseCreateCsrsv2Info(&info_Lc);
    // hipsparseCreateCsrsv2Info(&info_Uc);
    checkCudaErrors(cusparseStatus);

    // step 3: query how much memory used in csric02 and csrsv2, and allocate the buffer
    checkCudaErrors(hipsparseDcsric02_bufferSize(handle, 4, 6,
        descr_coeff, d_val, d_row, d_col, info_coeff_2, &pBufferSize_coeff));

    pBufferSize = pBufferSize_coeff; 

    checkCudaErrors(hipMalloc((void**)&pBuffer, pBufferSize));
    //(checkCudaErrors(cusparseStatus));


    // step 4: perform analysis of incomplete Cholesky on M
    //         perform analysis of triangular solve on L
    //         perform analysis of triangular solve on L'
    // The lower triangular part of M has the same sparsity pattern as L, so
    // we can do analysis of csric02 and csrsv2 simultaneously.

    checkCudaErrors(hipsparseDcsric02_analysis(handle, 4, 6,
        descr_coeff, d_val, d_row, d_col,info_coeff_2, 
        policy_coeff, pBuffer));

    cusparseStatus = hipsparseXcsric02_zeroPivot(handle, info_coeff_2, &structural_zero);
    if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus){
    printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    }

    // status = hipsparseXcsric02_zeroPivot(handle, info_M, &structural_zero);
    // if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
    // printf("A(%d,%d) is missing\n", structural_zero, structural_zero);
    // }
    //checkCudaErrors(cusparseStatus);


    //step 5: M = L * L'
    // checkCudaErrors(hipsparseDcsrilu02(handle, 4, 6,
    //     descr_coeff, d_val, d_row, d_col,info_coeff, 
    //     policy_coeff, pBuffer));
    // checkCudaErrors(cusparseStatus);

    // cusparseStatus = hipsparseXcsrilu02_zeroPivot(handle, info_coeff, &numerical_zero);
    // if (HIPSPARSE_STATUS_ZERO_PIVOT == cusparseStatus){
    //     printf("U(%d,%d) is zero\n", numerical_zero, numerical_zero);
    // }

    checkCudaErrors(hipMemcpy( J, d_col,6*sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy( I, d_row,5*sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy( val, d_val, 6*sizeof(double), hipMemcpyDeviceToHost));

    printf ("\n Matrix:\n");

    for (int i = 0; i< 6; i++){
        printf ("%f\t", val[i]);
    }



    
    printf ("\n Columns:\n");

    for (int i = 0; i < 6; i++){
        printf ("%d\t", J[i]);
    }

 
     printf ("\n Rows:\n");

    for (int i = 0; i < 5; i++){
        printf ("%d\t", I[i]);
    }
 
}




    // alpha = 1.0;
    // alpham1 = -1.0;
    // beta = 0.0;
    // r0 = 0.;

    // printf ("\n Matrix:\n");

    // for (int i = 0; i < N*M; i++){
    //     for (int j = 0; j< 5; j++){
    //         printf ("%f\t", val[i*5+j]);
    //     }

    //     printf("\n");
    // }

    
    // printf ("\n Columns:\n");

    // for (int i = 0; i < N*M; i++){
    //     for (int j = 0; j< 5; j++){
    //         printf ("%d\t", J[i*5+j]);
    //     }

    //     printf("\n");
    // }

    // printf ("\n Rows:\n");

    // for (int i = 0; i < N*M; i++){
        
    //     printf ("%d\n", I[i]);       

    //}
    // hipsparseSbsrmv(cusparseHandle,HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_x, &beta, d_Ax);

    // hipblasSaxpy(cublasHandle, N, &alpham1, d_Ax, 1, d_r, 1);
    // hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
       
    // k = 1;

    // while (r1 > tol*tol && k <= max_iter)
    // {
    //     if (k > 1)
    //     {
    //         b = r1 / r0;
    //         hipblasStatus_t = hipblasSscal(cublasHandle, N, &b, d_p, 1);
    //         hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &alpha, d_r, 1, d_p, 1);
    //     }
    //     else
    //     {
    //         hipblasStatus_t = hipblasScopy(cublasHandle, N, d_r, 1, d_p, 1);
    //     }

    //     hipsparseSbsrmv(cusparseHandle, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, nz, &alpha, descr, d_val, d_row, d_col, d_p, &beta, d_Ax);
    //     hipblasStatus_t = hipblasSdot(cublasHandle, N, d_p, 1, d_Ax, 1, &dot);
    //     a = r1 / dot;

    //     hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &a, d_p, 1, d_x, 1);
    //     na = -a;
    //     hipblasStatus_t = hipblasSaxpy(cublasHandle, N, &na, d_Ax, 1, d_r, 1);

    //     r0 = r1;
    //     hipblasStatus_t = hipblasSdot(cublasHandle, N, d_r, 1, d_r, 1, &r1);
    //     hipDeviceSynchronize();
    //     printf("iteration = %3d, residual = %e\n", k, sqrt(r1));
    //     k++;
    // }

    // hipMemcpy(x, d_x, N*sizeof(float), hipMemcpyDeviceToHost);

    // doublersum, diff, err = 0.0;

    // for (int i = 0; i < N; i++)
    // {
    //     rsum = 0.0;

    //     for (int j = I[i]; j < I[i+1]; j++)
    //     {
    //         rsum += val[j]*x[J[j]];
    //     }

    //     diff = fabs(rsum - rhs[i]);

    //     if (diff > err)
    //     {
    //         err = diff;
    //     }
    // }

    // hipsparseDestroy(cusparseHandle);
    // hipblasDestroy(cublasHandle);

    // free(I);
    // free(J);
    // free(val);
    // free(x);
    // free(rhs);
    // hipFree(d_col);
    // hipFree(d_row);
    // hipFree(d_val);
    // hipFree(d_x);
    // hipFree(d_r);
    // hipFree(d_p);
    // hipFree(d_Ax);

    // // hipDeviceReset causes the driver to clean up all state. While
    // // not mandatory in normal operation, it is good practice.  It is also
    // // needed to ensure correct operation when the application is being
    // // profiled. Calling hipDeviceReset causes all profile data to be
    // // flushed before the application exits
    // hipDeviceReset();

     //printf("Test Summary:  Error amount = %f\n", err);
    // exit((k <= max_iter) ? 0 : 1);
//}
