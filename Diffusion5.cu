#include "hip/hip_runtime.h"
/*Matrix A - define in CSR, convert to BSR*/
/*Vector b, vector r, Vector r-o*/
/*rho, alphs, pmega, itr - Scalars*/
//rho, i-1 and rho_i, omega_i-1 and omega_i
//Steps 
#include <stdlib.h>
#include <stdio.h>
#include <string.h>

/* Using updated (v2) interfaces to cublas */
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hipblas.h>

// Utilities and system includes
#include "helper_functions.h"  // helper for shared functions common to CUDA Samples
#include "hip/hip_runtime_api.h"       // helper function CUDA error checking and initialization
/*Initialize CSR*/
#include "cusparse_bcgstab_4.h"
#include "PostPr.h"
struct vertex {
    char VType; 
    char BSide; 
    char BType;
    double dely; 
    double delx; 
    double UValue; 
    double VValue; 
    double PValue; 
}; 

#define U0 1

__global__
void PRHSTry(double*RHS, int Nx, int Ny){

    int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

    if (thread_idx < Nx*Ny){
        RHS[thread_idx] = thread_idx%5*1.4;
    }
    
}

/*gernTriDiag generates a tridiogonal system of matrices in tridiagonal format
input I is the row vector, input J is the col vector, and val holds the values
Matrix generated is - [1 0 0.....0
                      -1 2 -1....0
                       0 -1 2 -1..0
                       .
                       .
                       .
                       0..-1 2 -1
                       0.....0...1]

Matrix is indexed in column-1 format
*/
void genTridiag(double *I, double *J, double*val, int rows, double mu, 
    double rho, double dely, double delx, double delT)
{



    //int start = 0;   //Holds the index for the vector in CSR//

    for (int i = 0; i < rows; i++)
    {
        I[i] = - mu/rho * (dely/delx)/(dely*delx) *delT/2;
        J[i] = - mu/rho * (dely/delx)/(dely*delx) *delT/2;
        val [i] = 1 + 2 * mu/rho * (dely/delx)/(dely*delx) *delT/2;
    }

    I[0] = 0;
    J[0] = 0;
    J[rows - 1] = 0;
    I[rows-1] = 0;
    val[rows-1] = 1;
}

/*Generates the RHS vector for linear solve
* All values are set to 1
*/
void genRHS (double* RHS, int rows, int Nx){
    for (int i=0; i < rows; i++){
        RHS[i] = (double)((i%Nx) * (i%Nx)) ;
    }
}

void genRHS2 (double* RHS, int rows, int Nx){
    for (int i=0; i < rows; i++){
        RHS[i] = (double)(rand()%10);
        if ( i >= 20 || i<5){
            RHS[i] = 0;
        }
    }
}


__global__
void XtoY(double*RHS, int Ny, int Nx, double*RHS_y){

    int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

    if (thread_idx < Nx*Ny){
        int row = thread_idx % Ny;
        int col =  thread_idx / Ny; 

        RHS_y[thread_idx] = RHS[row*Nx+col];
    }
    
}

__global__
void YtoX(double*RHS, int Ny, int Nx, double*RHS_y){

    int thread_idx = (blockIdx.x * blockDim.x) + threadIdx.x; 

    if (thread_idx < Nx*Ny){
        int row = thread_idx / Nx;
        int col =  thread_idx % Nx; 

        RHS_y[thread_idx] = RHS[col*Ny+row];
    }
    
}

/* Prints the results of the sparse solve 
* First prints the tridiagonal coefficent matrix
* Followed by the RHS and results*/

void printOutput(double *I, double *J, double*val, double* RHS, double* Soln, int rows){

    printf ("The coefficient matrix is : \n"); 
    printf ("\n");
     for (int i = 0; i < rows; i++){
        for (int j=0; j< rows; j++){
            if (j== (i-1)){

                printf ("%3.2f ", I[i]);

            } else if (j==i){
                printf ("%3.2f ", val[i]);
            } else if (j== (i+1)){
                printf ("%3.2f ", J[i]);
            } else {
                printf("0 "); 

            }


            }
            //valIndex ++;

            printf("\n");

        }

    printf ("\n The RHS vector is : \n");

    for (int i=0; i<rows; i++){
        printf ("%f\n", RHS[i]);
    }

    printf ("\n The solution is:\n");

    for (int i = 0; i < rows ; i++){
        
        printf ("%f\n", Soln[i]);       

    }
       
}

/* Solves a matrix system Ax=b
* Uses cusparse library
* A is represented in a CSR format
* I represent the row indices and J represents col indices
*/

float TriDiagSolve(double* low , double* diag , double* high, double* RHS, int rows){


    //int rows = 5; 
    int dimBlock = 1;   //CSR = BSR wth block dimension 1

    /*Allocate memory for host-side arrays*/


    /*Initialize the variables to be used in the linear solve*/
    hipsparseHandle_t handle = 0;
    hipsparseStatus_t cusparseStatus;
    cusparseStatus = hipsparseCreate(&handle);
    // hipsparseMatDescr_t descr_coeff = 0;
    // bsrsv2Info_t  info_coeff = 0;
    // const hipsparseSolvePolicy_t policy_coeff = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;
    // const hipsparseOperation_t trans_coeff  = HIPSPARSE_OPERATION_NON_TRANSPOSE;
    // const hipsparseDirection_t dir_coeff = HIPSPARSE_DIRECTION_ROW; //Doesn't matter for blockdIM = 1
    size_t pBufferSize; 
    void* pBuffer = 0;
    //double alpha = 1;
    int structural_zero, numerical_zero;    //To check for singularities in the coefficient matrix

    /*1 - Create descriptor for coeff matrix*/
    // checkCudaErrors(hipsparseCreateMatDescr(&descr_coeff));
    // checkCudaErrors(hipsparseSetMatIndexBase(descr_coeff, HIPSPARSE_INDEX_BASE_ONE));
    // checkCudaErrors(hipsparseSetMatType(descr_coeff, HIPSPARSE_MATRIX_TYPE_GENERAL));
    // hipsparseSetMatDiagType(descr_coeff, HIPSPARSE_DIAG_TYPE_NON_UNIT);

    // /*2 - Create info for linear solve*/
    // hipsparseCreateBsrsv2Info(&info_coeff);


    /*Create timer variables*/
    hipEvent_t startTime, stopTime; 
    float time; 
    hipEventCreate (&startTime);
    hipEventCreate (&stopTime);
    hipEventRecord (startTime, 0);


    /*4 - Allocate buffer space for linear solve*/
    checkCudaErrors(hipsparseDgtsv2_bufferSizeExt(handle, rows, 1, 
            low,diag, high, RHS ,rows , &pBufferSize));   //Why rows + 10?

    printf ("%zu", pBufferSize);
    checkCudaErrors(hipMalloc((void**)&pBuffer, sizeof(int)*pBufferSize));

    /*5 - Analyze coefficient matrix and report any singularities */
    checkCudaErrors(hipsparseDgtsv2(handle, rows, 1, 
            low,diag, high, RHS ,rows , pBuffer));

  

    /*Record end time*/
    hipEventRecord (stopTime, 0);
    hipEventSynchronize (stopTime);
    hipEventElapsedTime (&time, startTime, stopTime);
    hipEventDestroy (startTime);
    hipEventDestroy (stopTime); 

   



    return time;

}

/*Initializes input vectors X and Y for xuBlas
*/
void initializeVectors(double* X, double* Y, int rows){    

    for (int i=0; i < rows; i++){
        X[i] = (double)rand()/RAND_MAX; 
    }
    
    for (int i=0; i < rows; i++){
        Y[i] = 2*i%rows; 
    }
}

/*Adds two vectors, X and Y as Y = Y+X
* Uses cuBlas library
* Vector Y is overwritten
* Return : Execution time
*/

float callDiff (int timeStep, int Nx, int Ny){

    int rows = Nx * Ny; 

    double* I = (double *)malloc(sizeof(double)*(rows));
    double *J = (double *)malloc(sizeof(double)*rows);
    double *val_host = (double *)malloc(sizeof(double) * rows );
    double *RHS = (double *)malloc(sizeof(double) * rows);
    double *Soln = (double *)malloc(sizeof(double) * rows);
  
    double mu = 1E-5;
    double rho = 1.2;
    double delx = 0.001;
    double dely = 0.001; 
    /*Generate the sparse coefficient matrix*/
    //genTridiag (I, J, val_host, rows, mu, rho, dely, delx, delT);

    /*Generate RHS vector*/
    genRHS( RHS, rows, Nx); 

    double * I_dvc, *J_dvc; 
    double* val_dvc = NULL; 
    double *RHS_dvc = NULL; 
    double *Soln_Dvc = NULL; 
    double* Sol_y = NULL; 

    float Time; 

    int blockWidth = 128;

    /*Allocate memory on the device for the arrays*/
    checkCudaErrors(hipMalloc((double **)&J_dvc, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&I_dvc, (rows)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&val_dvc, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&RHS_dvc, (rows)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Soln_Dvc, (rows)*sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Sol_y, (rows)*sizeof(double)));



    /* 3 - Transfer data from host to device*/
    checkCudaErrors (hipMemcpy(J_dvc, J,  rows *sizeof(double), 
            hipMemcpyHostToDevice ));
    checkCudaErrors (hipMemcpy(val_dvc, val_host, rows *sizeof(double), 
            hipMemcpyHostToDevice ));
    checkCudaErrors (hipMemcpy(I_dvc, I, (rows)*sizeof(double),
             hipMemcpyHostToDevice ));
    checkCudaErrors (hipMemcpy( RHS_dvc, RHS,(rows)*sizeof(double), 
            hipMemcpyHostToDevice ));




    Time = TriDiagSolve(I_dvc , val_dvc, J_dvc, RHS_dvc, rows);

    int nBlocks = rows/blockWidth + 1; 

    XtoY<<<nBlocks, blockWidth>>> ( RHS_dvc, Ny, Nx, Sol_y);
     
     /* 7 - Transfer result data back to host*/
    checkCudaErrors (hipMemcpy( Soln, RHS_dvc,(rows)*sizeof(double), 
            hipMemcpyDeviceToHost));

    //Print output
    printOutput (I, J, val_host, RHS,Soln, rows);

    //double* Sol_y = (double *)malloc(sizeof(double) * rows);
    
    checkCudaErrors (hipMemcpy( Soln, Sol_y,(rows)*sizeof(double), 
            hipMemcpyDeviceToHost));

    printf ("\n The transposed vector is:\n");

    for (int i = 0; i < rows ; i++){
        
        printf ("%f\n", Soln[i]);       

    }

    TriDiagSolve(I_dvc , val_dvc, J_dvc, Sol_y, rows);

    checkCudaErrors (hipMemcpy( Soln, Sol_y,(rows)*sizeof(double), 
            hipMemcpyDeviceToHost));

    printf ("\n The solved vector is:\n");

    for (int i = 0; i < rows ; i++){
        
        printf ("%f\n", Soln[i]);       

    }

    YtoX<<<nBlocks, blockWidth>>> ( Sol_y, Ny, Nx, RHS_dvc);

    checkCudaErrors (hipMemcpy( Soln, RHS_dvc,(rows)*sizeof(double), 
            hipMemcpyDeviceToHost));

    printf ("\n The re-transposed vector is:\n");

    for (int i = 0; i < rows ; i++){
        
        printf ("%f\n", Soln[i]);       

    }


    //Free CUDA memory
    hipFree (I_dvc);
    hipFree (J_dvc);
    hipFree (val_dvc);
    hipFree (RHS_dvc);
    hipFree (Soln_Dvc);  


    //XtoY( Soln, rows, Ny, Nx, Sol_y); 



    return Time; 
}

/*Ver 2 - Added 2*/
__global__
void genXCoeffs(double *low, double *high, double*diag, const vertex* Domain, 
     int rows, double mu, double rho, double delT, char varID){

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    vertex thisV = Domain[thread_idx]; 
    double dely = thisV.dely; 
    double delx = thisV.delx; 

    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior vertices*/
                low [thread_idx] =  - (mu/rho * (dely/delx)/(dely*delx))*(delT/2);
                high [thread_idx] =  - mu/rho * (dely/delx)/(dely*delx) * (delT/2) ;
                diag [thread_idx] =  1 + 2 * mu/rho * (dely/delx)/(dely*delx) * (delT/2)  ;

                break;

            case '1':
                switch (thisV.BSide){
                    case 'L':
                    /*Left-Side Boundary*/
                        if ((thisV.BType == '0') || (thisV.BType == '2')) {
                            /*Dirichlet BC*/
                            low [thread_idx] =  0;
                            high [thread_idx] =  0;
                            diag [thread_idx] =  1 ;

                        } else if (thisV.BType == '1') {
                            /*Symmetry BC */
                            low [thread_idx] =  0;
                            high [thread_idx] =  - 1*mu/rho * (dely/delx)/(dely*delx)*(delT/2);
                            diag [thread_idx] =  1 + 1 * mu/rho * (dely/delx)/(dely*delx)* (delT/2);

                            
                            if (varID == 'U'){
                                high [thread_idx] = 0;
                                diag [thread_idx] = 1;
                            }

                        }
                    break;

                    case 'R':
                        /*Right-Side Boundary*/
                        if ((thisV.BType == '0')|| (thisV.BType == '2')){
                            /*Dirichlet BC*/
                            low [thread_idx] =  0;
                            high [thread_idx] =  0;
                            diag [thread_idx] =  1 ;

                        } else if (thisV.BType == '1') {
                            /*Symmetry BC */
                            high [thread_idx] =  0;
                            low [thread_idx] =  - 1*mu/rho * (dely/delx)/(dely*delx) * (delT/2);
                            diag [thread_idx] =  1 + 1 * mu/rho * (dely/delx)/(dely*delx) * (delT/2);

                            if (varID == 'U'){
                                low [thread_idx] = 0;
                                diag [thread_idx] = 1;
                            }

                        }

                    break; 
                    default:
                    /*Top and bottom boundaries treated as interior zones*/
                        low [thread_idx] =  - mu/rho * (dely/delx)/(dely*delx) * (delT/2);
                        high [thread_idx] =  - mu/rho * (dely/delx)/(dely*delx) * (delT/2);
                        diag [thread_idx] =  1 + 2 * mu/rho * (dely/delx)/(dely*delx) * (delT/2); 

                }

            break;
            case '2':

                if (thisV.BType == '0'){

                    /*Dirichlet condition*/
                     low [thread_idx] =  0;
                     high [thread_idx] =  0;
                     diag [thread_idx] =  1;

                } else{
                    if (thisV.BSide == 'W' || thisV.BSide == 'Z' ){

                    /*Left Side points*/
                        if (thisV.BType == '1') {

                                /*Symmetry BC*/
                                low [thread_idx] =  0;
                                high [thread_idx] =  - 2*mu/rho * (dely/delx)/(dely*delx) * (delT/2);
                                diag [thread_idx] =  1 + 2 * mu/rho * (dely/delx)/(dely*delx) * (delT/2);

                        }
                    } else if ( thisV.BSide == 'X' || thisV.BSide == 'Y') {
                        if (thisV.BType == '1') {

                                /*Symmetry BC*/
                                high [thread_idx] =  0;
                                low [thread_idx] =  - 2*mu/rho * (dely/delx)/(dely*delx) *(delT/2);
                                diag [thread_idx] =  1 + 2 * mu/rho * (dely/delx)/(dely*delx) * (delT/2);

                        }
                    }  
                }

            break;

        }

    }
}

/*Ver 2 -Added division by 2*/
__global__
void genYCoeffs(double *low, double *high, double*diag, const vertex* Domain, 
     int rows, double mu, double rho, double delT, char varID){

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    vertex thisV = Domain[thread_idx]; 
    double dely = thisV.dely; 
    double delx = thisV.delx; 

    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior vertices*/
                low [thread_idx] =  - mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                high [thread_idx] =  - mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                diag [thread_idx] =  1 + 2 * mu/rho * (delx/dely)/(dely*delx)  * (delT/2);

                break;

            case '1':
                switch (thisV.BSide){
                    case 'B':
                    /*Bottom-Side Boundary*/
                        if ((thisV.BType == '0') || (thisV.BType == '2')) {
                            /*Dirichlet BC*/
                            low [thread_idx] =  0;
                            high [thread_idx] =  0;
                            diag [thread_idx] =  1 ;

                        } else if (thisV.BType == '1') {
                            /*Symmetry BC */
                            high [thread_idx] =  0;
                            low [thread_idx] =  - 1*mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                            diag [thread_idx] =  1 + 1 * mu/rho * (delx/dely)/(dely*delx) * (delT/2);

                            if (varID == 'V'){
                                low [thread_idx] = 0;
                                diag [thread_idx] = 1;
                            }

                        }
                    break;

                    case 'T':
                        /*Top-Side Boundary*/
                        if ((thisV.BType == '0')|| (thisV.BType == '2')){
                            /*Dirichlet BC*/
                            low [thread_idx] =  0;
                            high [thread_idx] =  0;
                            diag [thread_idx] =  1 ;

                        } else if (thisV.BType == '1') {
                            /*Symmetry BC */
                            low [thread_idx] =  0;
                            high [thread_idx] =  - 1*mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                            diag [thread_idx] =  1 + 1 * mu/rho * (delx/dely)/(dely*delx) * (delT/2);

                            if (varID == 'V'){
                                high [thread_idx] = 0;
                                diag [thread_idx] = 1;
                            }

                        }

                    break; 
                    default:
                    /*Top and bottom boundaries treated as interior zones*/
                        low [thread_idx] =  - mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                        high [thread_idx] =  - mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                        diag [thread_idx] =  1 + 2 * mu/rho * (delx/dely)/(dely*delx) * (delT/2); 

                }

            break;
            case '2':

                if (thisV.BType == '0'){

                    /*Dirichlet condition*/
                     low [thread_idx] =  0;
                     high [thread_idx] =  0;
                     diag [thread_idx] =  1;

                } else{
                    if (thisV.BSide == 'X' || thisV.BSide == 'Z' ){

                    /*Bottom Side points*/
                        if (thisV.BType == '1') {

                                /*Symmetry BC*/
                                low [thread_idx] =  0;
                                high [thread_idx] =  - 2*mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                                diag [thread_idx] =  1 + 2 * mu/rho * (delx/dely)/(dely*delx) * (delT/2);

                        }
                    } else if ( thisV.BSide == 'D' || thisV.BSide == 'W') {

                        /*Top Side points*/

                        if (thisV.BType == '1') {

                                /*Symmetry BC*/
                                high [thread_idx] =  0;
                                low [thread_idx] =  - 2*mu/rho * (delx/dely)/(dely*delx) * (delT/2);
                                diag [thread_idx] =  1 + 2 * mu/rho * (delx/dely)/(dely*delx) * (delT/2);

                        }
                    }  
                }

            break;

        }

    }
}

void genYWithTrans(double *low, double *high, double*diag, const vertex* Domain, 
     int rows, double mu, double rho, double delT, int nBlocks, int blockWidth,
     int Ny, int Nx, hipblasHandle_t handleBlas, char varID){

        double* temp; 
        checkCudaErrors(hipMalloc((double **)&temp, rows *sizeof(double)));

        genYCoeffs <<<nBlocks,blockWidth>>> (low,high,diag,Domain, rows, mu, rho, delT, varID); 

        XtoY <<<nBlocks,blockWidth>>> (low,Ny, Nx, temp); 
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), temp, 1, low, 1));

        XtoY <<<nBlocks,blockWidth>>> (diag,Ny, Nx, temp); 
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), temp, 1, diag, 1));

        
        XtoY <<<nBlocks,blockWidth>>> (high,Ny, Nx, temp); 
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), temp, 1, high, 1));


     }
__global__
void calcH(const double *U, const double *V, const double*vec, double* H, 
    const vertex* Domain, int rows, int Nx){

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    vertex thisV = Domain[thread_idx]; 
    double dely = thisV.dely; 
    double delx = thisV.delx; 

    double right;
    double left;
    double top;
    double btm;

    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior vertices*/

                right = ((U[thread_idx]+ U[thread_idx+1])/2)*((vec[thread_idx]+ vec[thread_idx+1])/2);
                left = ((U[thread_idx]+ U[thread_idx-1])/2)*((vec[thread_idx]+ vec[thread_idx-1])/2);  
                top = ((V[thread_idx]+ V[thread_idx-Nx])/2)*((vec[thread_idx]+ vec[thread_idx-Nx])/2);
                btm = ((V[thread_idx]+ V[thread_idx+Nx])/2)*((vec[thread_idx]+ vec[thread_idx+Nx])/2);  
                H[thread_idx] = ((right - left)*dely + (top - btm )*delx)/(dely*delx);
                break;

            case '1':

                if (thisV.BType == '0' || thisV.BType == '2'){
                    /*Dirichlet condition*/

                    H[thread_idx] = 0;

                } else {
                    switch (thisV.BSide){
                        case 'B':
                            /*Bottom-Side Boundary*/
                            right = ((U[thread_idx]+ U[thread_idx+1])/2)*((vec[thread_idx]+ vec[thread_idx+1])/2);
                            left = ((U[thread_idx]+ U[thread_idx-1])/2)*((vec[thread_idx]+ vec[thread_idx-1])/2);  
                            H[thread_idx] = ((right - left)*dely)/(dely*delx);
                        break;

                        case 'T':
                            /*Top-Side Boundary*/
                            right = ((U[thread_idx]+ U[thread_idx+1])/2)*((vec[thread_idx]+ vec[thread_idx+1])/2);
                            left = ((U[thread_idx]+ U[thread_idx-1])/2)*((vec[thread_idx]+ vec[thread_idx-1])/2);  
                            H[thread_idx] = ((right - left)*dely)/(dely*delx);
                            break;

                        case 'R':
                        /*Right-Side Boundary*/
                            top = ((V[thread_idx]+ V[thread_idx-Nx])/2)*((vec[thread_idx]+ vec[thread_idx-Nx])/2);
                            btm = ((V[thread_idx]+ V[thread_idx+Nx])/2)*((vec[thread_idx]+ vec[thread_idx+Nx])/2);  
                            H[thread_idx] = ( (top - btm )*delx)/(dely*delx);
                        break;

                        case 'L':
                        /*Left-Side Boundary*/
                            top = ((V[thread_idx]+ V[thread_idx-Nx])/2)*((vec[thread_idx]+ vec[thread_idx-Nx])/2);
                            btm = ((V[thread_idx]+ V[thread_idx+Nx])/2)*((vec[thread_idx]+ vec[thread_idx+Nx])/2);  
                            H[thread_idx] = ( (top - btm )*delx)/(dely*delx);
                        break;

                    }

                break;
            }
            case '2':

                if (thisV.BType == '0' || thisV.BType == '2'){

                    /*Dirichlet condition*/
                    H[thread_idx] = 0;

                } else{
                     /*Symmetry condition*/
                    H[thread_idx] = 0;
                   
                }
            break; 
        }
    }
}


/*Has to be multiplied by dT/2 and added by u/v
1 + dT/2*D*/
__global__
void calcD(const double *vec, double*H,  const vertex* Domain, int rows, int Nx, 
    char varId, double delT, double mu, double rho){

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
    vertex thisV = Domain[thread_idx]; 
    double dely = thisV.dely; 
    double delx = thisV.delx; 
    double thisVar; 


    double right;
    double left;
    double top;
    double btm;

    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior vertices*/

                right = (vec[thread_idx+1]- vec[thread_idx])/delx;
                left = (vec[thread_idx]- vec[thread_idx-1])/delx;  
                top = (vec[thread_idx-Nx]- vec[thread_idx])/dely;
                btm = (vec[thread_idx]- vec[thread_idx + Nx])/dely; 
                H[thread_idx] = (((right - left)*dely + (top - btm )*delx)/(dely*delx))*delT/2 * mu/rho + vec[thread_idx];
                break;

            case '1':

                if (thisV.BType == '0' || thisV.BType == '2'){
                    /*Dirichlet condition*/
                    switch (varId){
                        case 'U':
                        H[thread_idx] = thisV.UValue;
                        break; 
                        case 'V':
                        H[thread_idx] = thisV.VValue;
                        break; 
                    }

                } else {
                    switch (thisV.BSide){
                        case 'B':
                            /*Bottom-Side Boundary*/

                            right = (vec[thread_idx+1]- vec[thread_idx])/delx;
                            left = (vec[thread_idx]- vec[thread_idx-1])/delx;  
                            top = (vec[thread_idx-Nx]- vec[thread_idx])/dely;
                            btm = (vec[thread_idx-Nx]- vec[thread_idx])/dely;
                            H[thread_idx] = ((right - left)*dely + (top - btm )*delx)/(dely*delx) * delT/2 * mu/rho + vec[thread_idx];
                        break;

                        case 'T':
                            /*Top-Side Boundary*/
                            right = (vec[thread_idx+1]- vec[thread_idx])/delx;
                            left = (vec[thread_idx]- vec[thread_idx-1])/delx;  
                            top = (vec[thread_idx]- vec[thread_idx + Nx])/dely;
                            btm = (vec[thread_idx]- vec[thread_idx + Nx])/dely;
                            H[thread_idx] = ((right - left)*dely + (top - btm )*delx)/(dely*delx) * delT/2 *mu/rho +vec[thread_idx];

                            break;

                        case 'R':
                        /*Right-Side Boundary*/
                            right = (vec[thread_idx]- vec[thread_idx-1])/delx;
                            left = (vec[thread_idx]- vec[thread_idx-1])/delx;  
                            top = (vec[thread_idx-Nx]- vec[thread_idx])/dely;
                            btm = (vec[thread_idx]- vec[thread_idx + Nx])/dely; 
                            H[thread_idx] = ((right - left)*dely + (top - btm )*delx)/(dely*delx) *delT/2 *mu/rho+vec[thread_idx];
                        break;

                        case 'L':
                        /*Left-Side Boundary*/
                            right = (vec[thread_idx+1]- vec[thread_idx])/delx;
                            left = (vec[thread_idx+1]- vec[thread_idx])/delx;  
                            top = (vec[thread_idx-Nx]- vec[thread_idx])/dely;
                            btm = (vec[thread_idx]- vec[thread_idx + Nx])/dely; 
                            H[thread_idx] = ((right - left)*dely + (top - btm )*delx)/(dely*delx) * delT/2 *mu/rho +vec[thread_idx] ;

                        break;

                    }

                break;
            }
            case '2':

                if (thisV.BType == '0' || thisV.BType == '2'){

                    /*Dirichlet condition*/
                    switch (varId){
                        case 'U':
                        H[thread_idx] = thisV.UValue;
                        break; 
                        case 'V':
                        H[thread_idx] = thisV.VValue;
                        break; 
                    }

                } else{
                     /*Symmetry condition*/
                    H[thread_idx] = 0;
                   
                }
            break; 
        }
    }
}

__global__
void Mesh (double Lx, double Ly, int Nx,int Ny, vertex* Domain){
    double y; 
    int rows = Ny*Nx; 
    int i = blockIdx.x * blockDim.x + threadIdx.x; 

    if (i < rows){
        if (i == 0){
            /* Top left corner*/
            Domain[i]. VType = '2';
            Domain[i].BSide = 'W';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(2*(Nx-1));
            Domain[i].BType = '0';
            Domain[i].UValue = 0;
            Domain[i].VValue = 0;
            Domain[i].PValue = 0;

        } else if (i== (Ny-1)){
            /*Top Right Corner*/
            Domain[i]. VType = '2';
            Domain[i].BSide = 'D';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(2*(Nx-1));
            Domain[i].BType = '0';
            Domain[i].UValue = U0;
            Domain[i].VValue = 0;
            Domain[i].PValue = 0;

        } else if (i==(Ny*(Nx-1))){
            /*Bottom Left Corner*/
            Domain[i]. VType = '2';
            Domain[i].BSide = 'Z';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(2*(Nx-1));
            Domain[i].BType = '0';
            Domain[i].UValue = 0;
            Domain[i].VValue = 0;
            Domain[i].PValue = 0;

        } else if (i== (rows -1)){
            /*Bottom Right Corner*/
            Domain[i]. VType = '2';
            Domain[i].BSide = 'X';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(2*(Nx-1));
            Domain[i].BType = '0';
            Domain[i].UValue = U0;
            Domain[i].VValue = 0;
            Domain[i].PValue = 0;

        } else if  (i < Ny){
            /*Top Edge*/
            Domain[i]. VType = '1';
            Domain[i].BSide = 'T';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(Nx-1);
            Domain[i].UValue = 0;
            Domain[i].VValue = 0;
            Domain[i].BType = '1';
            Domain[i].PValue = 0;

        } else if (i%Ny == 0){
            /*Left Edge*/
            Domain[i]. VType = '1';
            Domain[i].BSide = 'L';
            Domain[i]. dely = (double)Ly/(Ny-1);
            Domain[i].delx = (double) Lx/(2*(Nx-1));
            y = Ly - (i/Nx)* (double)Ly/(Ny-1); 
            Domain[i].UValue = U0 *6*(y-y*y) ;
            Domain[i].VValue = 0;
            Domain[i].BType = '0';
            Domain[i].PValue = 0;

        } else if (i%Ny == (Ny-1)){
            /*Right Edge*/
            Domain[i]. VType = '1';
            Domain[i].BSide = 'R';
            Domain[i]. dely = (double)Ly/(Ny-1);
            Domain[i].delx = (double)Lx/(2*(Nx-1));
            Domain[i].UValue = U0;
            Domain[i].VValue = 0;
            Domain[i].BType = '0';
            Domain[i].PValue = 0;

        } else if (i >= (Ny*(Nx-1)) ){
            /*Bottom Edge*/
            Domain[i]. VType = '1';
            Domain[i].BSide = 'B';
            Domain[i]. dely = (double)Ly/(2*(Ny-1));
            Domain[i].delx = (double)Lx/(Nx-1);
            Domain[i].UValue = 0;
            Domain[i].VValue = 0;
            Domain[i].BType = '1';
            Domain[i].PValue = 0;

        } else {
            /*Internal Points*/
            Domain[i]. VType = '0';
            Domain[i].BSide = '0';
            Domain[i].dely = (double)Ly/(Ny-1);
            Domain[i].delx = (double)Lx/(Nx-1);
            Domain[i].UValue = 0;
            Domain[i].VValue = 0;
            Domain[i].BType = '0';
            Domain[i].PValue = 0;
        } 
    }


      
}

void updateRHS(hipblasHandle_t handleBlas, double* H_u_n_1, double* H_v_n_1,
     double* H_u_n, double* H_v_n, double* D_u_n, double* D_v_n, 
     double* RHS_u_n, double* RHS_v_n, double delT, double mu, double rho, 
     int rows ){

    double alpha = -1.5 *delT; 

    /*Add vectors - Y = X + Y*/
    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                           &alpha,H_u_n_1, 1,RHS_u_n, 1));

    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                        &alpha,H_v_n_1, 1,RHS_v_n, 1));

    alpha = 0.5 *delT; 

    /*Add vectors - Y = X + Y*/
    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                           &alpha,H_u_n, 1,RHS_u_n, 1));

    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                        &alpha,H_v_n, 1, RHS_v_n, 1));


    alpha = 1; 

    /*Add vectors - Y = X + Y*/
    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                           &alpha,D_u_n, 1,RHS_u_n, 1));

    checkCudaErrors( hipblasDaxpy( handleBlas, rows, 
                        &alpha,D_v_n, 1,RHS_v_n, 1));  


    
}

/*Generates the coefficients of the pressure Poisson equation in CSR format*/
/*The basic pressure Poisson equation follows the pattern - 
*   [1 0 0 0.... 0 . . . .. . .]
     .
     .
     .1 .. 1  -4 1 ..1 0..0
     .
     .
     .0 ....0................1]


 *Stencil is [1,1,-4,1,1] and the variables are [Ptop, Pleft, P, Pright and Pbtm]
* Coeffieicnts are to be multiplied by the appropriate constsnat (Not done yet)
*/

__global__
void getPCoeffs(int* P_rowPtr, int* P_colPtr, double* P_val,
                 const vertex* Domain, int rows, int Nx, int Ny,
                 double rho, double delT){


    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x; 
    int start = thread_idx*5; 
    vertex thisV = Domain[thread_idx]; 
    double delx = thisV.delx;
    double dely = thisV.dely; 
    double constTerm  = 1/(rho*(delx*dely))*delT;


    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
                /*Interior vertices*/

                /*Coeff  = [1,1,-4,1,1]*/
                /*Variables = [Ptop, Pleft, P, Pright and Pbtm] */
                P_colPtr[start] = thread_idx - Nx; 
                P_colPtr[start+1] = thread_idx - 1; 
                P_colPtr[start+2] = thread_idx ; 
                P_colPtr[start+3] = thread_idx+1 ; 
                P_colPtr[start+4] = thread_idx+Nx ; 


                P_val[start] = (delx/dely)*constTerm; 
                P_val[start+1] = (dely/delx)*constTerm; 
                P_val[start+2] = -constTerm * (2*dely/delx + 2*delx/dely) ; 
                P_val[start+3] = (dely/delx)*constTerm ; 
                P_val[start+4] = (delx/dely)*constTerm;; 
                break;

            case '1':
                switch (thisV.BSide){
                    
                    case 'B' :
                    /*Bottom edge*/
                    /*Coeff  = [0,2,1,-4,1]*/
                    /*Var = [N/A, Ptop, Pleft, P, Pright]*/
                        P_colPtr[start] = thread_idx - (Ny-1)*Nx ; 
                        P_colPtr[start+1] = thread_idx - Nx; 
                        P_colPtr[start+2] = thread_idx - 1; 
                        P_colPtr[start+3] = thread_idx ; 
                        P_colPtr[start+4] = thread_idx+1 ; 
                       

                        P_val[start] = 0 ; 
                        P_val[start+1] = 2 * (delx/(2*dely))*constTerm; 
                        P_val[start+2] = (dely/delx)*constTerm; 
                        P_val[start+3] = -constTerm * (2*dely/delx + 2*delx/(2*dely)) ; 
                        P_val[start+4] = (dely/delx)*constTerm ; 
                       
                    break;  

                    case 'T' :
                    /*Top edge*/
                    /*Coeff  = [1,-4,1,2,0]*/
                    /*Var = [Pleft, P, Pright, Pbtm, N/A]*/
                        P_colPtr[start] = thread_idx - 1; 
                        P_colPtr[start+1] = thread_idx; 
                        P_colPtr[start+2] = thread_idx + 1; 
                        P_colPtr[start+3] = thread_idx + Nx; 
                        P_colPtr[start+4] = thread_idx+Nx*(Ny-1) ; 
                       

                        P_val[start] = (dely/delx)*constTerm ; 
                        P_val[start+1] = -constTerm * (2*dely/delx + 2*delx/(2*dely)); 
                        P_val[start+2] = (dely/delx)*constTerm; 
                        P_val[start+3] = 2 * (delx/(2*dely))*constTerm;
                        P_val[start+4] = 0 ; 
                       
                    break;

                    case 'L' :
                    /*Left Edge*/
                        P_colPtr[start] = thread_idx - Nx; 
                        P_colPtr[start+1] = thread_idx; 
                        P_colPtr[start+2] = thread_idx + 1; 
                        P_colPtr[start+3] = thread_idx + Nx-1; 
                        P_colPtr[start+4] = thread_idx+Nx ; 
                       

                        P_val[start] = (delx/(dely))*constTerm; 
                        P_val[start+1] =  -constTerm * (2*delx/dely + 2*dely/(2*delx)); 
                        P_val[start+2] = 2*(dely/(2*delx))*constTerm; 
                        P_val[start+3] = 0 ; 
                        P_val[start+4] = (delx/(dely))*constTerm;  
                       
                    break;

                    case 'R' :
                    /*Right edge*/
                    /*Outlet - Values are hard-coded to 0 for convergence*/
                        P_colPtr[start] = thread_idx - Nx; 
                        P_colPtr[start+1] = thread_idx-Nx+1; 
                        P_colPtr[start+2] = thread_idx - 1; 
                        P_colPtr[start+3] = thread_idx ; 
                        P_colPtr[start+4] = thread_idx+Nx; 
                       

                        // P_val[start] = 1 ; 
                        // P_val[start+1] = 0; 
                        // P_val[start+2] = 2; 
                        // P_val[start+3] = -4 ; 
                        // P_val[start+4] = 1 ; 

                        P_val[start] = 0 ; 
                        P_val[start+1] = 0; 
                        P_val[start+2] = 0; 
                        P_val[start+3] = 1 ; 
                        P_val[start+4] = 0 ;                       

                    break;

                }

            case '2':
                switch (thisV.BSide){       
                    case 'W':
                    /*Top left*/

                    P_colPtr[start] = thread_idx; 
                    P_colPtr[start+1] = thread_idx+1; 
                    P_colPtr[start+2] = thread_idx+Nx-1; 
                    P_colPtr[start+3] = thread_idx + Nx ; 
                    P_colPtr[start+4] = thread_idx+Nx*(Ny-1); 


                    P_val[start] = -constTerm * (2*delx/(2*dely) + 2*dely/(2*delx));  
                    P_val[start+1] = 2 * (dely/(2*delx))*constTerm; 
                    P_val[start+2] = 0; 
                    P_val[start+3] = 2 * (delx/(2*dely))*constTerm ; 
                    P_val[start+4] = 0 ; 
                    break;

                    case 'D':
                    /*Top right*/
                    P_colPtr[start] = thread_idx-Nx+1; 
                    P_colPtr[start+1] = thread_idx-1; 
                    P_colPtr[start+2] = thread_idx; 
                    P_colPtr[start+3] = thread_idx + Nx ; 
                    P_colPtr[start+4] = thread_idx+Nx*(Ny-1); 


                    P_val[start] = 0 ; 
                    P_val[start+1] = 0; 
                    P_val[start+2] = 1;  
                    P_val[start+3] = 0 ; 
                    P_val[start+4] = 0 ; 
                    break; 

                    case 'Z':
                    /*Bottom left*/

                    P_colPtr[start] = thread_idx-Nx*(Ny-1); 
                    P_colPtr[start+1] = thread_idx-Nx; 
                    P_colPtr[start+2] = thread_idx; 
                    P_colPtr[start+3] = thread_idx + 1 ; 
                    P_colPtr[start+4] = thread_idx+Nx-1; 


                    P_val[start] = 0 ; 
                    P_val[start+1] = 2 * (delx/(2*dely))*constTerm; 
                    P_val[start+2] = -constTerm * (2*delx/(2*dely) + 2*dely/(2*delx));  
                    P_val[start+3] = 2 * (dely/(2*delx))*constTerm ; 
                    P_val[start+4] = 0 ; 
                    break;
                    case 'X':
                    /*Bottom Right*/

                    P_colPtr[start] = thread_idx-(Ny-1)*Nx; 
                    P_colPtr[start+1] = thread_idx-Nx; 
                    P_colPtr[start+2] = thread_idx-Nx+1; 
                    P_colPtr[start+3] = thread_idx -1 ; 
                    P_colPtr[start+4] = thread_idx; 


                    P_val[start] = 0 ; 
                    P_val[start+1] = 0; 
                    P_val[start+2] = 0; 
                    P_val[start+3] = 0 ; 
                    P_val[start+4] = 1;   
                    break; 
                }

        }

        /*Increment row pointer by 5, as each element has stencil of 5*/
        P_rowPtr[thread_idx + 1] = thread_idx*5 + 5;
    }

}

/*Calculates the RHS for the pressure-poisson equation*/
/*The RHS is simple the divergence of vlocity
*/

__global__
void update_PRHS(double* P_RHS, double* U, double* V,
                 const vertex* Domain, int rows, int Nx, int Ny,
                 double rho){



    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x; 
    vertex thisV = Domain[thread_idx]; 
    double delx = thisV.delx;
    double dely = thisV.dely; 

    double rightV, leftV, topV, btmV; 


    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior vertices*/
            rightV = U[thread_idx+1];
            leftV = U[thread_idx-1];
            topV = V[thread_idx-Nx]; 
            btmV = V[thread_idx+Nx];

            P_RHS[thread_idx] = (rightV - leftV)/(2*delx) + (topV - btmV)/(2*dely);
            break;

            case '1':
                switch (thisV.BSide){
                    case 'B' :
                        /*Bottom edge*/
                        rightV = U[thread_idx+1];
                        leftV = U[thread_idx-1];

                        if ((thisV.BType == '0') || (thisV.BType == '2')){
                        /*Inlet or wall*/
                        /*Assumed that V = (Vbtm + Vtop)/2. So Vbtm = 2*V - Vtop*/
                        
                        topV = V[thread_idx-Nx];                       
                        btmV =  2*thisV.VValue - topV;   
                    } else {
                        /*Symmetry - no gradient*/
                        topV = 0;                       
                        btmV =  0;  
                    }

                    P_RHS[thread_idx] = (rightV - leftV)/(2*delx) + (topV - btmV)/(4*dely);                       
                    break;  

                    case 'T' :
                        /*Top Edge*/
                        rightV = U[thread_idx+1];
                        leftV = U[thread_idx-1];

                        if ((thisV.BType == '0') || (thisV.BType == '2')){
                        /*Inlet or wall*/
                        /*Assumed that V = (Vbtm + Vtop)/2. So Vtop = 2*V - Vbtm*/
                        btmV = V[thread_idx+Nx];                       
                        topV =  2*thisV.VValue - btmV; 
                        } else {
                        topV = 0;                       
                        btmV =  0;  
                    }

                    P_RHS[thread_idx] = (rightV - leftV)/(2*delx) + (topV - btmV)/(4*dely);                       
                    break;  

                    case 'L' :
                        /*Left edge*/
                        topV = V[thread_idx-Nx];
                        btmV = V[thread_idx + Nx]; 

                        if ((thisV.BType == '0') || (thisV.BType == '2')){
                        /*Inlet or wall*/
                        rightV = U[thread_idx+1];                       
                        leftV =  2*thisV.UValue - rightV; 
                        } else {
                        rightV = 0;                       
                        leftV =  0;  
                        }
                    
                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(2*dely);                       
                    break;  

                    case 'R' :
                    /*Right edge*/
                        topV = V[thread_idx-Nx];
                        btmV = V[thread_idx + Nx]; 

                        if ((thisV.BType == '0') || (thisV.BType == '2')){
                        /*Inlet or wall*/

                        leftV = U[thread_idx-1];                       
                        rightV =  2*thisV.UValue - leftV; 
                        } else {
                        rightV = 0;                       
                        leftV =  0;  
                        }
                    
                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(2*dely);    
                       
                    break;

                }

            case '2':
                switch (thisV.BSide){       
                    case 'W':
                    /*Top left*/

                    btmV = V[thread_idx + Nx]; 
                    rightV = U[thread_idx + 1]; 
                    topV = 2*thisV.VValue -  btmV; 
                    leftV = 2*thisV.UValue -  rightV; 

                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(4*dely);    
                    break;

                    case 'D':
                    /*Top right*/
                    btmV = V[thread_idx + Nx]; 
                    leftV = U[thread_idx - 1]; 
                    topV = 2*thisV.VValue - btmV; 
                    rightV = 2*thisV.UValue -  leftV; 

                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(4*dely);    
                    break;

                    case 'Z':
                    /*Bottom left*/

                    topV = V[thread_idx - Nx]; 
                    rightV = U[thread_idx + 1]; 
                    btmV = 2*thisV.VValue -  topV; 
                    leftV = 2*thisV.UValue -  rightV; 

                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(4*dely);    
                    break;

                    case 'X':
                    /*Bottom Right*/

                    topV = V[thread_idx - Nx]; 
                    leftV = U[thread_idx - 1]; 
                    btmV = 2*thisV.VValue -  topV; 
                    rightV = 2*thisV.UValue - leftV; 

                    P_RHS[thread_idx] = (rightV - leftV)/(4*delx) + (topV - btmV)/(4*dely);    
                    break;
 
                    break; 
                }

        }

    }

}

__global__
void velPressureCorrection (double*P, double* U, double* V,const vertex* Domain, 
        int rows, int Nx, int Ny,double rho, double delT ){

    int thread_idx = blockIdx.x * blockDim.x + threadIdx.x; 
    vertex thisV = Domain[thread_idx]; 
    double delx = thisV.delx;
    double dely = thisV.dely; 
    double constTerm = delT/(rho); 

    double right, left, top, btm; 

    if (thread_idx < rows){

        switch (thisV.VType){
            case '0':
            /*Interior Vertex*/
            right = P[thread_idx + 1];
            left = P[thread_idx - 1];
            top = P[thread_idx - Nx];
            btm = P[thread_idx + Nx];

            U[thread_idx] = U[thread_idx] - ( right - left)/(2*delx)*constTerm;
            V[thread_idx] = V[thread_idx] - (top-btm)/(2*dely)*constTerm;
            break; 

            case '1':
            /*Edge*/
            switch (thisV.BSide){

                case 'T':
                    /*Top Edge*/
                    if((thisV.VType == '0')||(thisV.VType == '2')){
                        top = 0;
                        btm = 0;
                        right = 0;
                        left = 0;
                    } else{
                        top = 0;                       
                        btm =  0; 
                        right = P[thread_idx + 1];
                        left = P[thread_idx - 1];                    
                    }

                    U[thread_idx] = U[thread_idx] - ( right - left)/(2*delx)*constTerm;
                    V[thread_idx] = V[thread_idx] - (top-btm)/(4*dely)*constTerm;
                    break; 

                case 'B':
                    /*Btm Edge*/


                    if((thisV.VType == '0')||(thisV.VType == '2')){
                        top = 0;
                        btm = 0;
                        right = 0;
                        left = 0; 
                    } else{
                        btm = 0;                       
                        top =  0; 
                        right = P[thread_idx + 1];
                        left = P[thread_idx - 1];
                    }

                    U[thread_idx] = U[thread_idx] - ( right - left)/(2*delx)*constTerm;
                    V[thread_idx] = V[thread_idx] - (top-btm)/(4*dely)*constTerm;
                    break;

                case 'R':
                    /*Right Edge*/


                    if((thisV.VType == '0')||(thisV.VType == '2')){
                        right = 0;
                        left = 0;
                        top = 0;
                        btm = 0;
                    } else{
                        left = 0;                       
                        right = 0; 
                        top = P[thread_idx - Nx];
                        btm = P[thread_idx + Nx];
                    } 

                    U[thread_idx] = U[thread_idx] - ( right - left)/(4*delx)*constTerm;
                    V[thread_idx] = V[thread_idx] - (top-btm)/(2*dely)*constTerm;
                    break;

                case 'L':
                    /*Left Edge*/


                    if((thisV.VType == '0')||(thisV.VType == '2')){
                        right = 0;
                        left = 0;
                        top = 0;
                        btm = 0;
                    } else{
                        right = 0;                       
                        left =  0; 
                        top = P[thread_idx - Nx];
                        btm = P[thread_idx + Nx];
                    } 

                    U[thread_idx] = U[thread_idx] - ( right - left)/(4*delx)*constTerm;
                    V[thread_idx] = V[thread_idx] - (top-btm)/(2*dely)*constTerm;
                    break;
            }


            break; 

            case '2':
                U[thread_idx] = U[thread_idx] ;
                V[thread_idx] = V[thread_idx] ;
                break; 

        }
    }
}


void Solve (double Lx, double Ly, int Nx,int Ny){

    int rows  = Ny*Nx; 
    int maxit = 40;
    vertex* DomainHost = (vertex*) malloc (sizeof(vertex)*rows);
    vertex* Domain = NULL; 
    double* Uxlow, * Uxhigh, * Uxdiag; 
    double* Vxlow, *Vxhigh, *Vxdiag; 
    double* Uylow, * Uyhigh, * Uydiag; 
    double* Vylow, *Vyhigh, *Vydiag; 
    double* U, *V, *P; 
    double* H_u_n_1, *H_v_n_1, *H_u_n, *H_v_n; 
    double* D_u_n, *D_v_n;
    double* RHS_u_n, *RHS_v_n;
    int* P_rowPtr, *P_colPtr;  
    double* P_val, *P_RHS; 

    double mu = 1;
    double rho = 1;
    double delT =0.001;
    int nSteps = 51;
    int nRecordedSteps = nSteps%10 + 1; 
    
  

    //double* I = (double *)malloc(sizeof(double)*(rows));
    //double *J = (double *)malloc(sizeof(double)*rows);
    //double *val_host = (double *)malloc(sizeof(double) * rows );
    double *U_Result= (double *)malloc(sizeof(double) * rows * nRecordedSteps );
    double *V_Result= (double *)malloc(sizeof(double) * rows * nRecordedSteps );
    //double *Soln = (double *)malloc(sizeof(double) * rows);
    //double*temp = (double *)malloc(sizeof(double) * 5*rows);
    //int* temp2 = (int*)malloc(sizeof(int) * 5*rows);
    //int* temp3 = (int*)malloc(sizeof(int) * (rows+1));



    hipblasHandle_t handleBlas = 0;
    hipblasStatus_t hipblasStatus_t;
    hipblasStatus_t = hipblasCreate(&handleBlas);
    checkCudaErrors (hipblasStatus_t);


    checkCudaErrors(hipMalloc((vertex **)&Domain, rows *sizeof(vertex)));

    checkCudaErrors(hipMalloc((double **)&Uxlow, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Uxhigh, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Uxdiag, rows *sizeof(double)));

    
    checkCudaErrors(hipMalloc((double **)&Uylow, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Uyhigh, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Uydiag, rows *sizeof(double)));

    checkCudaErrors(hipMalloc((double **)&Vxlow, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Vxhigh, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Vxdiag, rows *sizeof(double)));

    checkCudaErrors(hipMalloc((double **)&Vylow, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Vyhigh, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&Vydiag, rows *sizeof(double)));

    checkCudaErrors(hipMalloc((double **)&U, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&V, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&P, rows *sizeof(double)));

    checkCudaErrors(hipMalloc((double **)&H_u_n_1, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&H_v_n_1, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&H_u_n, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&H_v_n, rows *sizeof(double)));
    
    checkCudaErrors(hipMalloc((double **)&D_u_n, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&D_v_n, rows *sizeof(double)));


    checkCudaErrors(hipMalloc((double **)&RHS_u_n, rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&RHS_v_n, rows *sizeof(double)));

    checkCudaErrors(hipMalloc((double **)&P_rowPtr, (rows+1) *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&P_colPtr, 5*rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&P_val, 5*rows *sizeof(double)));
    checkCudaErrors(hipMalloc((double **)&P_RHS, rows *sizeof(double)));

    int blockWidth = 128;
    int nBlocks = rows/blockWidth + 1; 



    Mesh <<<nBlocks,blockWidth>>> (Lx,Ly,Nx,Ny, Domain);
    
    getPCoeffs<<<nBlocks,blockWidth>>>(P_rowPtr, P_colPtr, P_val,
             Domain,  rows, Nx, Ny, rho, delT);

    genXCoeffs <<<nBlocks,blockWidth>>> (Uxlow,Uxhigh,Uxdiag,Domain, rows, mu, rho, delT, 'U'); 
    genXCoeffs <<<nBlocks,blockWidth>>> (Vxlow,Vxhigh,Vxdiag,Domain, rows, mu, rho, delT, 'V'); 
    genYWithTrans (Uylow,Uyhigh,Uydiag,Domain, rows, mu, rho, delT, nBlocks, blockWidth,
        Ny, Nx, handleBlas, 'U' ); 
    genYWithTrans (Vylow,Vyhigh,Vydiag,Domain, rows, mu, rho, delT, nBlocks, blockWidth,
        Ny, Nx, handleBlas, 'V' ); 

    for (int t = 0; t<nSteps; t++){

        calcH <<<nBlocks,blockWidth>>> (U,V,U, H_u_n, Domain, rows, Nx);
        calcH <<<nBlocks,blockWidth>>> (U,V,V, H_v_n, Domain, rows, Nx);
        calcD <<<nBlocks,blockWidth>>> (U,D_u_n, Domain, rows, Nx, 'U', delT, mu,rho );
        calcD <<<nBlocks,blockWidth>>> (V,D_v_n, Domain, rows, Nx, 'V', delT, mu, rho);

        updateRHS (handleBlas, H_u_n_1,  H_v_n_1,
        H_u_n,  H_v_n,  D_u_n, D_v_n, 
        RHS_u_n, RHS_v_n, delT, mu, rho, 
        rows); 



        TriDiagSolve(Uxlow , Uxdiag, Uxhigh, RHS_u_n, rows);
        //checkCudaErrors(hipblasDcopy(handleBlas,(rows), RHS_u_n, 1, U, 1));
        XtoY<<<nBlocks, blockWidth>>> ( RHS_u_n, Ny, Nx, U);
        TriDiagSolve(Uylow , Uydiag, Uyhigh, U, rows);
        YtoX<<<nBlocks, blockWidth>>> ( U, Ny, Nx, RHS_u_n);
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), RHS_u_n, 1, U, 1));


        TriDiagSolve(Vxlow , Vxdiag, Vxhigh, RHS_v_n, rows);
        //checkCudaErrors(hipblasDcopy(handleBlas,(rows), RHS_u_n, 1, U, 1));
        XtoY<<<nBlocks, blockWidth>>> ( RHS_v_n, Ny, Nx, V);
        TriDiagSolve(Vylow , Vydiag, Vyhigh, V, rows);
        YtoX<<<nBlocks, blockWidth>>> ( V, Ny, Nx, RHS_v_n);
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), RHS_v_n, 1, V, 1));


        update_PRHS <<<nBlocks, blockWidth>>> ( P_RHS, U, V, Domain, rows, Nx, Ny,
                 rho); 


        LinearSolve( P_rowPtr, P_colPtr, P_val, 
                    P, P_RHS,                   
                    rows, 5*rows, maxit );


        velPressureCorrection <<<nBlocks, blockWidth>>>(P, U, V, Domain, 
            rows,  Nx, Ny, rho,  delT ); 

        checkCudaErrors(hipblasDcopy(handleBlas,(rows), H_u_n, 1, H_u_n_1, 1));
        checkCudaErrors(hipblasDcopy(handleBlas,(rows), H_v_n, 1, H_v_n_1, 1));
        checkCudaErrors(hipMemset(RHS_u_n, 0, rows*sizeof(double)));
        checkCudaErrors(hipMemset(RHS_v_n, 0, rows*sizeof(double)));

        if (t%10 ==0){
        

            checkCudaErrors (hipMemcpy( &U_Result[t/10], U,(rows)*sizeof(double), 
                hipMemcpyDeviceToHost));

            checkCudaErrors (hipMemcpy( &V_Result[t/10], V,(rows)*sizeof(double), 
                hipMemcpyDeviceToHost));

            
        }

        // if (t %10 ==0){

        //     printf ("\n Pressure Corrected V\n");

        //     for (int i = 0; i < rows; i++){

        //         printf("%f\n", RHS[i]);
        //     }
        // }
    }

    writeOutOutFile (U_Result, rows, nPrintedSteps, 'U'); 
    writeOutOutFile (U_Result, rows, nPrintedSteps, 'V'); 


}

    
 
    
/* Starting point of the program
* Creates a linear system of a problem size from cmd prompts
* Solves the system using cuSparse
* Adds two vectors of problemSize using cuBlas
* 
*/

int main(int argc, char** argv){
    float time; 
    int Ny = 50;
    int Nx = 50;

    if (argc >= 2) {
            Ny = atoi(argv[1]);
            Nx = atoi(argv[2]);
    }

    // validate command line arguments
    if (Ny< 3) {
        
        Ny = 3;
        printf("Warning: Problem size can't be less than 3\n");
        printf("The total number of threads will be modified  to 3\n");
    }


    Solve(1,1,Nx,Ny);

    printf ("\n\nSolving  linear  equations using cuSparse library\n"); 
    //time = callDiff( 1,Nx, Ny);
    printf ("The time taken for linear solve is \n");
    //printf ("%3.1f mus", time );

    // printf ("\n\nAdding two vectors using cuBlas library\n");
    // time = vectorAdd (rows);  
    // printf ("The time taken for vector add is \n");
    // printf ("%3.1f mus", time );
}
